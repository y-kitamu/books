#include <stdio.h>
#include <hip/hip_runtime.h>



constexpr int N = 1024;
constexpr int threadsPerBlock = 512;

__global__ void gpu_dot(float *d_a, float *d_b, float *d_c) {
    __shared__ float partial_sum[threadsPerBlock];
    int tid = threadIdx.x + blockIdx.x * blockDim.x;
    int index = threadIdx.x;

    // printf("d_a[%d] = %.0f\n", tid, d_a[tid]);
    float sum = 0;
    while (tid < N) {
        sum += d_a[tid] * d_b[tid];
        tid += blockDim.x * gridDim.x;
    }

    partial_sum[index] = sum;
    __syncthreads();

    int i = blockDim.x / 2;
    while (i != 0) {
        if (index < i) {
            partial_sum[index] += partial_sum[index + i];
        }
        __syncthreads();
        i /= 2;
    }

    if (index == 0) {
        d_c[blockIdx.x] = partial_sum[0];
    }
}


int main() {
    float *h_a, *h_b, h_c, *partial_sum;
    float *d_a, *d_b, *d_partial_sum;

    int block_calc = (N + threadsPerBlock - 1) / threadsPerBlock;

    int blocksPerGrid = (32 < block_calc ? 32 : block_calc);

    h_a = (float*)malloc(N * sizeof(float));
    h_b = (float*)malloc(N * sizeof(float));
    partial_sum = (float*)malloc(N * sizeof(float));

    hipMalloc((void**)&d_a, N * sizeof(float));
    hipMalloc((void**)&d_b, N * sizeof(float));
    hipMalloc((void**)&d_partial_sum, N * sizeof(float));

    for (int i = 0; i < N; i++) {
        h_a[i] = i;
        h_b[i] = 2;
    }

    hipMemcpy(d_a, h_a, sizeof(float) * N, hipMemcpyHostToDevice);
    hipMemcpy(d_b, h_b, sizeof(float) * N, hipMemcpyHostToDevice);
    printf("blocksPerGrid = %d\n", blocksPerGrid);
    gpu_dot<<<blocksPerGrid, threadsPerBlock>>>(d_a, d_b, d_partial_sum);
    hipDeviceSynchronize();
    hipMemcpy(partial_sum, d_partial_sum, sizeof(float) * N, hipMemcpyDeviceToHost);

    h_c = 0;
    for (int i = 0; i < blocksPerGrid; i++) {
        h_c += partial_sum[i];
    }

    printf("The computed dot product is : %f\n", h_c);

    if ((int)h_c == (N * (N - 1))) {
        printf("The dot product computed by GPU is correct\n");
    } else {
        printf("Error in dot product computation\n");
    }

    hipFree(d_a);
    hipFree(d_b);
    hipFree(d_partial_sum);

    free(h_a);
    free(h_b);
    free(partial_sum);
}