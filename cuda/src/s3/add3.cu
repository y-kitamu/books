/*
 * add3.cpp
 *
 * Create Date : 2021-06-04 10:45:32
 * Copyright (c) 2019- Yusuke Kitamura <ymyk6602@gmail.com>
 */

#include <hip/hip_runtime.h>
#include <stdio.h>

constexpr int N = 5000;

__global__ void gpuAdd(int* d_a, int* d_b, int* d_c) {
    int tid = threadIdx.x + blockIdx.x * blockDim.x;

    while (tid < N) {
        d_c[tid] = d_a[tid] + d_b[tid];
        tid += blockDim.x * gridDim.x;
    }
}

int main() {
    int h_a[N], h_b[N], h_c[N];
    int *d_a, *d_b, *d_c;

    hipMalloc((void**)&d_a, N * sizeof(int));
    hipMalloc((void**)&d_b, N * sizeof(int));
    hipMalloc((void**)&d_c, N * sizeof(int));

    for (int i = 0; i < N; i++)  {
        h_a[i] = 2 * i * i;
        h_b[i] = i;
    }

    hipMemcpy(d_a, h_a, N * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_b, h_b, N * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_c, h_c, N * sizeof(int), hipMemcpyHostToDevice);

    gpuAdd<<<512, 512>>>(d_a, d_b, d_c);
    hipMemcpy(h_c, d_c, N * sizeof(int), hipMemcpyDeviceToHost);

    hipDeviceSynchronize();

    int Correct = 1;
    for (int i = 0; i < N; i++) {
        if ((h_a[i] + h_b[i] != h_c[i])) {
            Correct = 0;
        }
    }
    if (Correct == 1) {
        printf("Gpu has computed Sum Correctry\n");
    } else {
        printf("There is an error in GPU Computation\n");
    }

    hipFree(d_a);
    hipFree(d_b);
    hipFree(d_c);
}
