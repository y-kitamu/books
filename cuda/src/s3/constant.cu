#include <stdio.h>


#include <hip/hip_runtime.h>



__constant__ int constant_f;
__constant__ int constant_g;

constexpr int N = 5;


__global__ void gpu_constant_memory(float *d_in, float *d_out) {
    int tid = threadIdx.x;
    d_out[tid] = constant_f * d_in[tid] + constant_g;
}


int main () {
    float h_in[N], h_out[N];
    float *d_in, *d_out;

    int h_f = 2;
    int h_g = 20;

    hipMalloc((void**)&d_in, N * sizeof(float));
    hipMalloc((void**)&d_out, N * sizeof(float));

    for (int i = 0; i < N; i++) {
        h_in[i] = i;
    }

    hipMemcpy(d_in, h_in, N * sizeof(float), hipMemcpyHostToDevice);

    hipMemcpyToSymbol(HIP_SYMBOL(constant_f), &h_f, sizeof(int), 0, hipMemcpyHostToDevice);
    hipMemcpyToSymbol(HIP_SYMBOL(constant_g), &h_g, sizeof(int));

    gpu_constant_memory<<<1, N>>>(d_in, d_out);

    hipMemcpy(h_out, d_out, N * sizeof(float), hipMemcpyDeviceToHost);

    printf("Use of Constant memory on GPU\n");
    for (int i = 0; i < N; i++) {
        printf("The expression for index %f is %f\n", h_in[i], h_out[i]);
    }

    hipFree(d_in);
    hipFree(d_out);
    return 0;
}