#include <stdio.h>

#include <hip/hip_runtime.h>


constexpr int N = 100000;


__global__ void gpuAdd(int *d_a, int *d_b, int *d_c) {
    int tid = blockDim.x * blockIdx.x + threadIdx.x;
    while (tid < N) {
        d_c[tid] = d_a[tid] + d_b[tid];
        tid += gridDim.x * blockDim.x;
    }
}


int main() {
    int h_a[N], h_b[N], h_c[N];
    int *d_a, *d_b, *d_c;

    for (int i = 0; i < N; i++) {
        h_a[i] = i;
        h_b[i] = i * 2;
    }

    hipEvent_t e_start, e_stop;
    hipEventCreate(&e_start);
    hipEventCreate(&e_stop);

    hipEventRecord(e_start, 0);

    hipMalloc((void**)&d_a, N * sizeof(int));
    hipMalloc((void**)&d_b, N * sizeof(int));
    hipMalloc((void**)&d_c, N * sizeof(int));

    hipMemcpy(d_a, h_a, N * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_b, h_b, N * sizeof(int), hipMemcpyHostToDevice);

    gpuAdd<<<512, 512>>>(d_a, d_b, d_c);

    hipMemcpy(h_c, d_c, N * sizeof(int), hipMemcpyHostToDevice);

    hipDeviceSynchronize();
    hipEventRecord(e_stop, 0);
    hipEventSynchronize(e_stop);

    float elapsedTime;
    hipEventElapsedTime(&elapsedTime, e_start, e_stop);
    printf("Time to add %d numbers : %3.1f ms\n", N, elapsedTime);
}
