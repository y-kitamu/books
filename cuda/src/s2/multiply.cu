
#include <hip/hip_runtime.h>
#include <stdio.h>


__global__ void multiply(int* d_a, int* d_b, int* d_c) {
    *d_c = (*d_a) * (*d_b);
}


int main() {
    int h_a, h_b, h_c;
    int *d_a, *d_b, *d_c;

    h_a = 10;
    h_b = 20;

    hipMalloc((void**)(&d_a), sizeof(int));
    hipMalloc((void**)(&d_b), sizeof(int));
    hipMalloc((void**)(&d_c), sizeof(int));

    hipMemcpy(d_a, &h_a, sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_b, &h_b, sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_c, &h_c, sizeof(int), hipMemcpyHostToDevice);

    multiply<<<500, 10>>>(d_a, d_b, d_c);
    multiply<<<10, 500>>>(d_a, d_b, d_c);
    multiply<<<50, 100>>>(d_a, d_b, d_c);

    hipMemcpy(&h_c, d_c, sizeof(int), hipMemcpyDeviceToHost);

    printf("%d * %d = %d\n", h_a, h_b, h_c);

    hipFree(d_a);
    hipFree(d_b);
    hipFree(d_c);
}
