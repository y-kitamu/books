
#include <hip/hip_runtime.h>
#include <stdio.h>

constexpr int N = 50;

__global__ void cube(int* d_in, int* d_out) {
    int tid = threadIdx.x;
    if (tid < N) {
        int tmp = d_in[tid];
        d_out[tid] = tmp * tmp * tmp;
    }
}


int main() {
    int h_in[N], h_out[N];
    int *d_in, *d_out;

    for (int i = 0; i < N; i++) {
        h_in[i] = i;
    }

    hipMalloc((void**)(&d_in), sizeof(int) * 50);
    hipMalloc((void**)(&d_out), sizeof(int) * 50);

    hipMemcpy(d_in, &h_in, sizeof(int) * N, hipMemcpyHostToDevice);

    cube<<<1, N>>>(d_in, d_out);

    hipMemcpy(&h_out, d_out, sizeof(int) * N, hipMemcpyDeviceToHost);

    for (int i = 0; i < N; i++) {
        printf("cube of %d is %d\n", h_in[i], h_out[i]);
    }

    hipFree(d_in);
    hipFree(d_out);
}
