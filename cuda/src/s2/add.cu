#include <iostream>
#include <hip/hip_runtime.h>



__global__ void gpuAdd(int *d_a, int *d_b, int* d_c) {
    *d_c = *d_a + *d_b;
    printf("%p\n", d_c);
}

int main() {
    int h_a, h_b, h_c;
    int *d_a, *d_b, *d_c;

    printf("%p\n", &h_a);
    printf("%p\n", &h_b);
    printf("%p\n", &h_c);
    printf("%p\n", &d_a);
    printf("%p\n", &d_b);
    printf("%p\n", &d_c);

    h_a = 1;
    h_b = 4;

    hipMalloc((void**)&d_a, sizeof(int));
    hipMalloc((void**)&d_b, sizeof(int));
    hipMalloc((void**)&d_c, sizeof(int));

    hipMemcpy(d_a, &h_a, sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_b, &h_b, sizeof(int), hipMemcpyHostToDevice);

    gpuAdd<<<1, 10>>>(d_a, d_b, d_c);
    hipMemcpy(&h_c, d_c, sizeof(int), hipMemcpyDeviceToHost);

    printf("%d + %d = %d\n", h_a, h_b, h_c);
    hipFree(d_a);
    hipFree(d_b);
    hipFree(d_c);
    return 0;
}