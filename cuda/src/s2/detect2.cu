#include <stdio.h>
#include <hip/hip_runtime.h>


int main() {
    int device_count = 0;
    hipGetDeviceCount(&device_count);
    for (int device = 0; device < device_count; device++) {
        hipDeviceProp_t device_property;
        hipGetDeviceProperties(&device_property, device);

        memset(&device_property, 0, sizeof(device_property));
        device_property.major = 5;
        device_property.minor = 0;

        int dev;
        hipChooseDevice(&dev, &device_property);
        printf("Id of device which version is 5.0 or greater : %d\n", dev);
    }
}
